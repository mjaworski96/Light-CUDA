#include "hip/hip_runtime.h"
#include "image_program.cuh"

#include <iostream>

#include "../kernel.cuh"
#include "../graphic/image.cuh"
#include "../graphic/pixels_factory.cuh"
#include "../graphic/rgb_pixels_factory.cuh"

int ImageProgram::main(int argc, char * argv[])
{
	if (argc != 6)
	{
		std::cout << "Invalid parameters count" << std::endl;
		usage();
		return -2;
	}
	srand(time(NULL));
	int cols = atoi(argv[2]);
	int rows = atoi(argv[3]);
	int count = atoi(argv[4]);;

	PixelsFactory* factory = new RgbPixelsFactory();
	Image image(rows, cols);
	int size = image.getSize();

	int blockSize = 256;
	int numBlocks = ceil((size + blockSize - 1) / (1.0 * blockSize));

	Pixel* inputPixels = factory->generate(count, cols, rows);
	Color* deviceResult;
	Pixel* deviceInputPixels;

	hipMallocManaged(&deviceResult, size * sizeof(Color));
	hipMallocManaged(&deviceInputPixels, count * sizeof(Pixel));
	hipMemcpy(deviceInputPixels, inputPixels, count * sizeof(Pixel), hipMemcpyHostToDevice);

	generateImage << < numBlocks, blockSize >> >
		(deviceInputPixels, count,
			deviceResult, cols, rows, size);
	hipDeviceSynchronize();
	Color* result = new Color[size];
	hipMemcpy(result, deviceResult, size * sizeof(Color), hipMemcpyDeviceToHost);
	image.replacePixels(result);
	image.save(argv[5]);


	hipFree(deviceInputPixels);
	hipFree(deviceResult);

	delete[] result;
	delete[] inputPixels;
	return 0;
}

void ImageProgram::usage()
{
	std::cout << "program -i [cols] [rows] [count] [image] " << std::endl;
}
