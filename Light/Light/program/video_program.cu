#include "hip/hip_runtime.h"
#include "video_program.cuh"

#include <iostream>

#include "../kernel.cuh"
#include "../graphic/video.cuh"
#include "../graphic/pixels_factory.cuh"
#include "../graphic/rgb_pixels_factory.cuh"

void VideoProgram::movePixels(MovablePixel * pixels, int count, double step)
{
	for (int i = 0; i < count; i++)
	{
		pixels[i].move(step);
	}
}

MovablePixel * VideoProgram::createMovablePixels(Pixel * pixels, int count, int maxX, int maxY, int maxSpeed)
{
	MovablePixel* movable = new MovablePixel[count];

	for (int i = 0; i < count; i++)
	{
		movable[i].init(&pixels[i], maxX, maxY, maxSpeed);
	}

	return movable;
}

void VideoProgram::simulate(MovablePixel* pixels, int count, double step, double gravityConstans)
{
	for (int i = 0; i < count; i++)
	{
		unsigned char lightness1 = pixels[i].pixel->color.getLightness();
		if (lightness1 == 0)
		{
			continue;
		}
		for (int j = i + 1; j < count; j++)
		{
			unsigned char lightness2 = pixels[j].pixel->color.getLightness();
			if (lightness2 == 0)
			{
				continue;
			}
			double dx = pixels[j].pixel->point.x - pixels[i].pixel->point.x;
			double dy = pixels[j].pixel->point.y - pixels[i].pixel->point.y;
			double distanceSquare = dx * dx + dy * dy;
			if (distanceSquare > 0)
			{
				Point2D move(dx, dy);
				pixels[i].vector = pixels[i].vector + (move * (step * gravityConstans * lightness2 * lightness2 / distanceSquare));
				pixels[j].vector = pixels[j].vector + (move * (-step * gravityConstans * lightness1 * lightness1 / distanceSquare));
			}
		}
	}
}


int VideoProgram::main(int argc, char * argv[])
{
	if (argc != 10)
	{
		std::cout << "Invalid parameters count" << std::endl;
		usage();
		return -1;
	}
	srand(time(NULL));
	int cols = atoi(argv[2]);
	int rows = atoi(argv[3]);
	int time = atoi(argv[4]);;
	int fps = atoi(argv[5]);;
	int maxSpeed = atoi(argv[6]);
	int count = atoi(argv[7]);
	double gravity = atof(argv[8]);
	Video video(argv[9], fps, cols, rows);



	double step = 1.0 / fps;
	int framesCount = fps * time;

	PixelsFactory* factory = new RgbPixelsFactory();
	Image image(rows, cols);
	int size = image.getSize();

	int blockSize = 256;
	int numBlocks = ceil((size + blockSize - 1) / (1.0 * blockSize));


	Pixel* inputPixels = factory->generate(count, cols, rows);
	MovablePixel* movable = createMovablePixels(inputPixels, count, cols, rows, maxSpeed);
	Color* deviceResult;
	Pixel* deviceInputPixels;


	for (int i = 0; i < framesCount; i++)
	{
		std::cout << "Frame: " << i + 1 << "/" << framesCount << " " << (i + 1) * 100 / framesCount << "%" << std::endl;

		hipMallocManaged(&deviceResult, size * sizeof(Color));
		hipMallocManaged(&deviceInputPixels, count * sizeof(Pixel));
		hipMemcpy(deviceInputPixels, inputPixels, count * sizeof(Pixel), hipMemcpyHostToDevice);

		generateImage << < numBlocks, blockSize >> >
			(deviceInputPixels, count,
				deviceResult, cols, rows, size);
		hipDeviceSynchronize();

		Color* result = new Color[size];
		hipMemcpy(result, deviceResult, size * sizeof(Color), hipMemcpyDeviceToHost);

		image.replacePixels(result);
		delete[] result;

		video.addFrame(image);
		simulate(movable, count, step, gravity);
		movePixels(movable, count, step);

		hipFree(deviceInputPixels);
		hipFree(deviceResult);
	}

	video.finish();
	delete[] movable;
	delete[] inputPixels;
	return 0;
}

void VideoProgram::usage()
{
	std::cout << "program -v [cols] [rows] [time] [fps] [maxSpeed] [input pixels count] [gravity] [video]" << std::endl;
}
