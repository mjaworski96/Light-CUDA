#include "hip/hip_runtime.h"
#include "rgb_pixels_factory.cuh"

int RgbPixelsFactory::random(int max)
{
	return rand() % max;
}
unsigned char RgbPixelsFactory::randomChar()
{
	return random(256);
}

Pixel * RgbPixelsFactory::generate(int count, int maxX, int maxY)
{
	Pixel* pixels = new Pixel[count];

	for (int i = 0; i < count; i++)
	{
		pixels[i].color.red = randomChar();
		pixels[i].color.green = randomChar();
		pixels[i].color.blue = randomChar();
		pixels[i].point.x = random(maxX);
		pixels[i].point.y = random(maxY);
	}
	

	return pixels;
}