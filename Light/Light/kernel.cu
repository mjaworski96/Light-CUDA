#include "hip/hip_runtime.h"
#include "kernel.cuh"


__device__
unsigned char normalizeColor(int sum)
{
	if (sum > 255) return 255;
	return (unsigned char)sum;
}
__global__
void generateImage(Pixel* pixels, int pixelsCount,
	Color* result, int cols, int rows, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < size; i += stride)
	{
		int currentRow = index / cols;
		int currentCol = index % cols;
		int red = 0, green = 0, blue = 0;
		for (int j = 0; j < pixelsCount; j++)
		{
			double rowDist = currentRow - pixels[j].point.y;
			double colDist = currentCol - pixels[j].point.x;
			double distance = rowDist * rowDist + colDist * colDist + 1;
			// abs(rowDist) + abs(colDist) + 1;
			red += pixels[j].color.red * pixels[j].color.red / distance;
			green += pixels[j].color.green * pixels[j].color.green / distance;
			blue += pixels[j].color.blue * pixels[j].color.blue / distance;
		}
		result[i].red = normalizeColor(red);
		result[i].green = normalizeColor(green);
		result[i].blue = normalizeColor(blue);
	}
}